
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
 #define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/
    __shared__ float partialSum[BLOCK_SIZE*4];
    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;
    if(t<=size){
        partialSum[t] = in[start + t];
        partialSum[blockDim.x+t] = in[start + blockDim.x+t];
    }else{
        partialSum[t]=0;
    }
     __syncthreads();
     for (unsigned int stride = 1;stride <= blockDim.x; stride *= 2)
        {
        __syncthreads();
        if (t % stride == 0) partialSum[2*t]+= partialSum[2*t+stride];
        }
     __syncthreads();
   
     if (t == 0) out[blockIdx.x] = partialSum[0];
}

__device__ void warpReduce(volatile float* sdata, int tid) {
sdata[tid] += sdata[tid + 32];
sdata[tid] += sdata[tid + 16];
sdata[tid] += sdata[tid + 8];
sdata[tid] += sdata[tid + 4];
sdata[tid] += sdata[tid + 2];
sdata[tid] += sdata[tid + 1];
}


__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
    __shared__ float partialSum[BLOCK_SIZE*4];
    unsigned int t = threadIdx.x;
    unsigned int start = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    partialSum[t] = in[start] + in[start+blockDim.x];

     __syncthreads();
    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
       if (t < s) partialSum[t] += partialSum[t + s];
       __syncthreads();
    }
    if (t < 32) warpReduce(partialSum, t);
   
    if (t == 0) out[blockIdx.x] = partialSum[0];
}
